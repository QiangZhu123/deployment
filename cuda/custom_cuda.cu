#include "custom_cuda_kernel.cuh"
#include "pytorch_cuda_helper.hpp"

/*
void CustomForwardCUDAKernelLauncher(Tensor input ,Tensor output){
简化的函数调用
custom_forward_cuda_kernel<float><<<THREADS_PER_BLOCK, THREADS_PER_BLOCK>>>(
        input.data_ptr<float>(),output.data_ptr<float>());

};
*/
void CustomForwardCUDAKernelLauncher(Tensor input ,Tensor output){

    //计算需要的线程数量
  int output_size = output.numel();
  int channels = input.size(1);
  int height = input.size(2);
  int width = input.size(3);
  
  at::cuda::CUDAGuard device_guard(input.device());
  hipStream_t stream = at::cuda::getCurrentCUDAStream();
  
  AT_DISPATCH_FLOATING_TYPES_AND_HALF(
      input.scalar_type(), "custom_forward_cuda_kernel", [&] {
        custom_forward_cuda_kernel<scalar_t><<<GET_BLOCKS(output_size), THREADS_PER_BLOCK, 0, stream>>>(
                input.data_ptr<scalar_t>(),output.data_ptr<scalar_t>());
      });

  AT_CUDA_CHECK(hipGetLastError()); 

};



void CustomBackwardCUDAKernelLauncher(Tensor input ,Tensor grad_output){

  int output_size = grad_output.numel();
  int channels = input.size(1);
  int height = input.size(2);
  int width = input.size(3);
  
  at::cuda::CUDAGuard device_guard(grad_output.device());
  hipStream_t stream = at::cuda::getCurrentCUDAStream();
  AT_DISPATCH_FLOATING_TYPES_AND_HALF(
      grad_output.scalar_type(), "custom_backward_cuda_kernel", [&] {
        custom_backward_cuda_kernel<scalar_t>
            <<<GET_BLOCKS(output_size), THREADS_PER_BLOCK, 0, stream>>>(
                grad_output.data_ptr<scalar_t>(),
                input.data_ptr<scalar_t>());
      });

  AT_CUDA_CHECK(hipGetLastError());
  
};
